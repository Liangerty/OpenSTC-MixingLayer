#include "hip/hip_runtime.h"
#include "DPLUR.cuh"
#include "BoundCond.cuh"

namespace cfd {
__global__ void convert_dq_back_to_dqDt(DZone *zone, const DParameter *param) {
  const int extent[3]{zone->mx, zone->my, zone->mz};
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  const int j = blockDim.y * blockIdx.y + threadIdx.y;
  const int k = blockDim.z * blockIdx.z + threadIdx.z;
  if (i >= extent[0] || j >= extent[1] || k >= extent[2]) return;

  const real dt_local = zone->dt_local(i, j, k);
  auto &dq = zone->dq;
  for (int l = 0; l < param->n_var; ++l) {
    dq(i, j, k, l) /= dt_local;
  }
}

__global__ void set_dq_to_0(const DParameter *param, DZone *zone, int i_face) {
  const auto &b = zone->boundary[i_face];
  auto range_start = b.range_start, range_end = b.range_end;
  int i = range_start[0] + (int) (blockDim.x * blockIdx.x + threadIdx.x);
  int j = range_start[1] + (int) (blockDim.y * blockIdx.y + threadIdx.y);
  int k = range_start[2] + (int) (blockDim.z * blockIdx.z + threadIdx.z);
  if (i > range_end[0] || j > range_end[1] || k > range_end[2]) return;

  for (int l = 0; l < param->n_var; ++l) {
    zone->dq(i, j, k, l) = 0;
  }
}

void set_wall_dq_to_0(const Block &block, const DParameter *param, DZone *zone, const DBoundCond &bound_cond) {
  for (size_t l = 0; l < bound_cond.n_wall; l++) {
    const auto nb = bound_cond.wall_info[l].n_boundary;
    for (size_t i = 0; i < nb; i++) {
      auto [i_zone, i_face] = bound_cond.wall_info[l].boundary[i];
      if (i_zone != block.block_id) {
        continue;
      }
      const auto &hf = block.boundary[i_face];
      const auto ngg = block.ngg;
      uint tpb[3], bpg[3];
      for (size_t j = 0; j < 3; j++) {
        const auto n_point = hf.range_end[j] - hf.range_start[j] + 1;
        tpb[j] = n_point <= (2 * ngg + 1) ? 1 : 16;
        bpg[j] = (n_point - 1) / tpb[j] + 1;
      }
      dim3 TPB{tpb[0], tpb[1], tpb[2]}, BPG{bpg[0], bpg[1], bpg[2]};
      set_dq_to_0<<<BPG, TPB>>>(param, zone, i_face);
    }
  }
}

}