#include "hip/hip_runtime.h"
#include "Driver.cuh"
#include "Initialize.cuh"
#include "DataCommunication.cuh"
#include "TimeAdvanceFunc.cuh"
#include "WallDistance.cuh"
#include "MixingLayer.cuh"
#include "SpongeLayer.cuh"

namespace cfd {

template<MixtureModel mix_model, class turb>
Driver<mix_model, turb>::Driver(Parameter &parameter, Mesh &mesh_):
    myid(parameter.get_int("myid")), time(), mesh(mesh_), parameter(parameter),
    spec(parameter), reac(parameter, spec), flameletLib(parameter), stat_collector(parameter, mesh, field) {
  // Allocate the memory for every block
  parameter.deduce_sim_info(spec);

  if (myid == 0)
    printf("\n*****************************Driver initialization******************************\n");

  for (int blk = 0; blk < mesh.n_block; ++blk) {
    field.emplace_back(parameter, mesh[blk]);
  }

  if (parameter.get_int("initial") == 1) {
    // If continue from previous results, then we need the residual scales
    // If the file does not exist, then we have a trouble
    std::ifstream res_scale_in("output/message/residual_scale.txt");
    res_scale_in >> res_scale[0] >> res_scale[1] >> res_scale[2] >> res_scale[3];
    res_scale_in.close();
  }

  for (int blk = 0; blk < mesh.n_block; ++blk) {
    field[blk].setup_device_memory(parameter);
  }
  printf("\tProcess [[%d]] has finished setting up device memory.\n", myid);
  bound_cond.initialize_bc_on_GPU(mesh_, field, spec, parameter);

  initialize_basic_variables<mix_model, turb>(parameter, mesh, field, spec);

  if (parameter.get_bool("sponge_layer")) {
    initialize_sponge_layer(parameter, mesh, field, spec);
  }

  write_reference_state(parameter, spec);

  DParameter d_param(parameter, spec, &reac, &flameletLib);
  hipMalloc(&param, sizeof(DParameter));
  hipMemcpy(param, &d_param, sizeof(DParameter), hipMemcpyHostToDevice);

  if (parameter.get_bool("steady") == 0 && parameter.get_bool("if_collect_statistics")) {
    stat_collector.initialize_statistics_collector<mix_model, turb>(spec);
  }
}

template<MixtureModel mix_model, class turb>
void Driver<mix_model, turb>::initialize_computation() {
  dim3 tpb{8, 8, 4};
  if (mesh.dimension == 2) {
    tpb = {16, 16, 1};
  }
  const auto ng_1 = 2 * mesh[0].ngg - 1;

  if (myid == 0)
    printf("\n******************************Prepare to compute********************************\n");

  // If we use k-omega SST model, we need the wall distance, thus we need to compute or read it here.
  if constexpr (TurbMethod<turb>::needWallDistance == true) {
    // SST method
    acquire_wall_distance<mix_model, turb>(*this);
  } else {
    if (parameter.get_int("if_compute_wall_distance") == 1) {
      acquire_wall_distance<mix_model, turb>(*this);
    }
  }

  if (mesh.dimension == 2) {
    for (auto b = 0; b < mesh.n_block; ++b) {
      const auto mx{mesh[b].mx}, my{mesh[b].my};
      dim3 BPG{(mx + ng_1) / tpb.x + 1, (my + ng_1) / tpb.y + 1, 1};
      eliminate_k_gradient <<<BPG, tpb >>>(field[b].d_ptr, param);
    }
  }

  // Second, apply boundary conditions to all boundaries, including face communication between faces
  for (int b = 0; b < mesh.n_block; ++b) {
    bound_cond.apply_boundary_conditions<mix_model, turb>(mesh[b], field[b], param);
  }
  printf("\tProcess [[%d]] has finished applying boundary conditions for initialization\n", myid);


  // First, compute the conservative variables from basic variables
  if constexpr (TurbMethod<turb>::hasMut == true) {
    for (auto i = 0; i < mesh.n_block; ++i) {
      int mx{mesh[i].mx}, my{mesh[i].my}, mz{mesh[i].mz};
      dim3 bpg{(mx + ng_1) / tpb.x + 1, (my + ng_1) / tpb.y + 1, (mz + ng_1) / tpb.z + 1};
      initialize_mut<mix_model, turb><<<bpg, tpb>>>(field[i].d_ptr, param);
    }
  }
  hipDeviceSynchronize();
  // Third, communicate values between processes
  printf("\tProcess [[%d]] is going to transfer data\n", myid);
  data_communication<mix_model, turb>(mesh, field, parameter, 0, param);
  printf("\tProcess [[%d]] has finished data transfer\n", myid);
  hipDeviceSynchronize();

  for (auto b = 0; b < mesh.n_block; ++b) {
    int mx{mesh[b].mx}, my{mesh[b].my}, mz{mesh[b].mz};
    dim3 bpg{(mx + ng_1) / tpb.x + 1, (my + ng_1) / tpb.y + 1, (mz + ng_1) / tpb.z + 1};
    update_physical_properties<mix_model><<<bpg, tpb>>>(field[b].d_ptr, param);
  }
  hipDeviceSynchronize();
  if (myid == 0) {
    printf("\tThe driver is completely initialized on GPU.\n");
  }
}

__global__ void compute_wall_distance(const real *wall_point_coor, DZone *zone, int n_point_times3) {
  const int ngg{zone->ngg}, mx{zone->mx}, my{zone->my}, mz{zone->mz};
  int i = (int) (blockDim.x * blockIdx.x + threadIdx.x) - ngg;
  int j = (int) (blockDim.y * blockIdx.y + threadIdx.y) - ngg;
  int k = (int) (blockDim.z * blockIdx.z + threadIdx.z) - ngg;
  if (i >= mx + ngg || j >= my + ngg || k >= mz + ngg) return;

  const real x{zone->x(i, j, k)}, y{zone->y(i, j, k)}, z{zone->z(i, j, k)};
  const int n_wall_point = n_point_times3 / 3;
  auto &wall_dist = zone->wall_distance(i, j, k);
  wall_dist = 1e+6;
  for (int l = 0; l < n_wall_point; ++l) {
    const int idx = 3 * l;
    real d = (x - wall_point_coor[idx]) * (x - wall_point_coor[idx]) +
             (y - wall_point_coor[idx + 1]) * (y - wall_point_coor[idx + 1]) +
             (z - wall_point_coor[idx + 2]) * (z - wall_point_coor[idx + 2]);
    if (wall_dist > d) {
      wall_dist = d;
    }
  }
  wall_dist = std::sqrt(wall_dist);
}

void write_reference_state(Parameter &parameter, const Species &species) {
  if (parameter.get_int("myid") == 0) {
    printf("\n*******************************Flow Information*********************************\n");
    printf("\tReference state:\n");
    std::filesystem::path out_dir("output/message");
    if (!exists(out_dir)) {
      create_directories(out_dir);
    }
    FILE *ref_state = fopen("output/message/reference_state.txt", "w");
    if (parameter.get_int("problem_type") == 1) {
      // For mixing layers, we need to output info about both streams.
      std::vector<real> var_info;
      cfd::get_mixing_layer_info(parameter, species, var_info);

      printf("\tUpper stream\n");
      printf("\t\t->-> %-16.10e : density(kg/m3)\n", var_info[0]);
      printf("\t\t->-> %-16.10e : u(m/s)\n", var_info[1]);
      printf("\t\t->-> %-16.10e : v(m/s)\n", var_info[2]);
      printf("\t\t->-> %-16.10e : w(m/s)\n", var_info[3]);
      real u1{std::sqrt(var_info[1] * var_info[1] + var_info[2] * var_info[2] + var_info[3] * var_info[3])};
      printf("\t\t->-> %-16.10e : velocity(m/s)\n", u1);
      printf("\t\t->-> %-16.10e : pressure(Pa)\n", var_info[4]);
      printf("\t\t->-> %-16.10e : temperature(K)\n", var_info[5]);

      fprintf(ref_state, "Upper stream:\ndensity = %16.10e\n", var_info[0]);
      fprintf(ref_state, "u = %16.10e\n", var_info[1]);
      fprintf(ref_state, "v = %16.10e\n", var_info[2]);
      fprintf(ref_state, "w = %16.10e\n", var_info[3]);
      fprintf(ref_state, "velocity = %16.10e\n", u1);
      fprintf(ref_state, "pressure = %16.10e\n", var_info[4]);
      fprintf(ref_state, "temperature = %16.10e\n", var_info[5]);

      int ns{species.n_spec};
      real mu, gamma{gamma_air}, mw{mw_air};
      if (ns > 0) {
        real cp_i[MAX_SPEC_NUMBER];
        species.compute_cp(var_info[5], cp_i);
        real cp{0};
        mw = 0;
        for (const auto &[name, i]: species.spec_list) {
          if (var_info[6 + i] > 0) {
            printf("\t\t->-> %-16.10e : Y_%s\n", var_info[6 + i], name.c_str());
            fprintf(ref_state, "Y_%s = %16.10e\n", name.c_str(), var_info[6 + i]);
          }
          mw += var_info[6 + i] / species.mw[i];
          cp += cp_i[i] * var_info[6 + i];
        }
        gamma = cp / (cp - R_u * mw);
        mw = 1 / mw;
        mu = compute_viscosity(var_info[5], mw, &var_info[6], species);
      } else {
        mu = Sutherland(var_info[5]);
      }
      real c1{std::sqrt(gamma * R_u / mw * var_info[5])};

      printf("\t\t->-> %-16.10e : speed_of_sound(m/s)\n", c1);
      printf("\t\t->-> %-16.10e : specific_heat_ratio\n", gamma);
      printf("\t\t->-> %-16.10e : Ma\n", u1 / c1);
      printf("\t\t->-> %-16.10e : Re_unit(/m)\n", var_info[0] * u1 / mu);
      printf("\t\t->-> %-16.10e : mu(kg/m/s)\n", mu);

      fprintf(ref_state, "speed_of_sound = %16.10e\n", c1);
      fprintf(ref_state, "specific_heat_ratio = %16.10e\n", gamma);
      fprintf(ref_state, "Ma = %16.10e\n", u1 / c1);
      fprintf(ref_state, "Re_unit = %16.10e\n", var_info[0] * u1 / mu);
      fprintf(ref_state, "mu = %16.10e\n", mu);

      // Next, the lower stream
      printf("\tLower stream\n");
      printf("\t\t->-> %-16.10e : density(kg/m3)\n", var_info[7 + ns]);
      printf("\t\t->-> %-16.10e : u(m/s)\n", var_info[8 + ns]);
      printf("\t\t->-> %-16.10e : v(m/s)\n", var_info[9 + ns]);
      printf("\t\t->-> %-16.10e : w(m/s)\n", var_info[10 + ns]);
      real u2{std::sqrt(var_info[8 + ns] * var_info[8 + ns] + var_info[9 + ns] * var_info[9 + ns] +
                        var_info[10 + ns] * var_info[10 + ns])};
      printf("\t\t->-> %-16.10e : velocity(m/s)\n", u2);
      printf("\t\t->-> %-16.10e : pressure(Pa)\n", var_info[11 + ns]);
      printf("\t\t->-> %-16.10e : temperature(K)\n", var_info[12 + ns]);

      fprintf(ref_state, "\nLower stream:\ndensity = %16.10e\n", var_info[7 + ns]);
      fprintf(ref_state, "u = %16.10e\n", var_info[8 + ns]);
      fprintf(ref_state, "v = %16.10e\n", var_info[9 + ns]);
      fprintf(ref_state, "w = %16.10e\n", var_info[10 + ns]);
      fprintf(ref_state, "velocity = %16.10e\n", u2);
      fprintf(ref_state, "pressure = %16.10e\n", var_info[11 + ns]);
      fprintf(ref_state, "temperature = %16.10e\n", var_info[12 + ns]);
      if (ns > 0) {
        real cp_i[MAX_SPEC_NUMBER];
        species.compute_cp(var_info[12 + ns], cp_i);
        real cp{0};
        mw = 0;
        for (const auto &[name, i]: species.spec_list) {
          if (var_info[13 + ns + i] > 0) {
            printf("\t\t->-> %-16.10e : Y_%s\n", var_info[13 + ns + i], name.c_str());
            fprintf(ref_state, "Y_%s = %16.10e\n", name.c_str(), var_info[13 + ns + i]);
          }
          mw += var_info[13 + ns + i] / species.mw[i];
          cp += cp_i[i] * var_info[13 + ns + i];
        }
        gamma = cp / (cp - R_u * mw);
        mw = 1 / mw;
        mu = compute_viscosity(var_info[12 + ns], mw, &var_info[13 + ns], species);
      } else {
        mu = Sutherland(var_info[12 + ns]);
      }

      real c2 = std::sqrt(gamma * R_u / mw * var_info[12 + ns]);
      printf("\t\t->-> %-16.10e : speed_of_sound(m/s)\n", c2);
      printf("\t\t->-> %-16.10e : specific_heat_ratio\n", gamma);
      printf("\t\t->-> %-16.10e : Ma\n", u2 / c2);
      printf("\t\t->-> %-16.10e : Re_unit(/m)\n", var_info[7 + ns] * u2 / mu);
      printf("\t\t->-> %-16.10e : mu(kg/m/s)\n", mu);

      fprintf(ref_state, "speed_of_sound = %16.10e\n", c2);
      fprintf(ref_state, "specific_heat_ratio = %16.10e\n", gamma);
      fprintf(ref_state, "Ma = %16.10e\n", u2 / c2);
      fprintf(ref_state, "Re_unit = %16.10e\n", var_info[7 + ns] * u2 / mu);
      fprintf(ref_state, "mu = %16.10e\n", mu);

      // Compute the convective velocity
      real uc = (u1 * c2 + u2 * c1) / (c1 + c2);
      parameter.update_parameter("convective_velocity", uc);
      printf("\n\t\t->-> %-16.10e : convective velocity(m/s)\n", uc);
      fprintf(ref_state, "convective velocity = %16.10e\n", uc);
      // Velocity ratio and density ratio
      real density_ratio = var_info[0] / var_info[7 + ns];
      real velocity_ratio = u1 / u2;
      printf("\t\t->-> %-16.10e : density_ratio\n", density_ratio);
      printf("\t\t->-> %-16.10e : velocity_ratio\n", velocity_ratio);
      fprintf(ref_state, "density_ratio = %16.10e\n", density_ratio);
      fprintf(ref_state, "velocity_ratio = %16.10e\n", velocity_ratio);
      parameter.update_parameter("density_ratio", density_ratio);
      parameter.update_parameter("velocity_ratio", velocity_ratio);
      // Compute the velocity delta
      real DeltaU = abs(u1 - u2);
      parameter.update_parameter("DeltaU", DeltaU);
      printf("\t\t->-> %-16.10e : DeltaU\n", DeltaU);
      fprintf(ref_state, "DeltaU = %16.10e\n", DeltaU);
    } else {
      printf("\t\t->-> %-16.10e : density(kg/m3)\n", parameter.get_real("rho_inf"));
      printf("\t\t->-> %-16.10e : velocity(m/s)\n", parameter.get_real("v_inf"));
      printf("\t\t->-> %-16.10e : u(m/s)\n", parameter.get_real("ux_inf"));
      printf("\t\t->-> %-16.10e : v(m/s)\n", parameter.get_real("uy_inf"));
      printf("\t\t->-> %-16.10e : pressure(Pa)\n", parameter.get_real("p_inf"));
      printf("\t\t->-> %-16.10e : temperature(K)\n", parameter.get_real("T_inf"));
      auto &sv_ref = parameter.get_real_array("sv_inf");
      for (const auto &[name, i]: species.spec_list) {
        if (sv_ref[i] > 0)
          printf("\t\t->-> %-16.10e : Y_%s\n", sv_ref[i], name.c_str());
      }
      printf("\t\t->-> %-16.10e : Ma\n", parameter.get_real("M_inf"));
      printf("\t\t->-> %-16.10e : Re_unit(/m)\n", parameter.get_real("Re_unit"));
      printf("\t\t->-> %-16.10e : mu(kg/m/s)\n", parameter.get_real("mu_inf"));
      printf("\t\t->-> %-16.10e : acoustic_speed(m/s)\n", parameter.get_real("speed_of_sound"));
      printf("\t\t->-> %-16.10e : specific_heat_ratio\n", parameter.get_real("specific_heat_ratio_inf"));

      fprintf(ref_state, "Reference state\nrho_ref = %16.10e\n", parameter.get_real("rho_inf"));
      fprintf(ref_state, "v_ref = %16.10e\n", parameter.get_real("v_inf"));
      fprintf(ref_state, "p_ref = %16.10e\n", parameter.get_real("p_inf"));
      fprintf(ref_state, "T_ref = %16.10e\n", parameter.get_real("T_inf"));
      for (const auto &[name, i]: species.spec_list) {
        if (sv_ref[i] > 0)
          fprintf(ref_state, "Y_%s = %16.10e\n", name.c_str(), sv_ref[i]);
      }
      fprintf(ref_state, "Ma_ref = %16.10e\n", parameter.get_real("M_inf"));
      fprintf(ref_state, "Re_unit = %16.10e\n", parameter.get_real("Re_unit"));
      fprintf(ref_state, "mu_ref = %16.10e\n", parameter.get_real("mu_inf"));
      fprintf(ref_state, "acoustic_speed_ref = %16.10e\n", parameter.get_real("speed_of_sound"));
      fprintf(ref_state, "specific_heat_ratio = %16.10e\n", parameter.get_real("specific_heat_ratio_inf"));
    }
    fclose(ref_state);
  }
}

// Instantiate all possible drivers
template
struct Driver<MixtureModel::Air, Laminar>;
template
struct Driver<MixtureModel::Air, SST<TurbSimLevel::RANS>>;
template
struct Driver<MixtureModel::Air, SST<TurbSimLevel::DES>>;
template
struct Driver<MixtureModel::Mixture, Laminar>;
template
struct Driver<MixtureModel::Mixture, SST<TurbSimLevel::RANS>>;
template
struct Driver<MixtureModel::Mixture, SST<TurbSimLevel::DES>>;
template
struct Driver<MixtureModel::FR, Laminar>;
template
struct Driver<MixtureModel::FR, SST<TurbSimLevel::RANS>>;
template
struct Driver<MixtureModel::FR, SST<TurbSimLevel::DES>>;
template
struct Driver<MixtureModel::MixtureFraction, Laminar>;
template
struct Driver<MixtureModel::MixtureFraction, SST<TurbSimLevel::RANS>>;
template
struct Driver<MixtureModel::MixtureFraction, SST<TurbSimLevel::DES>>;
template
struct Driver<MixtureModel::FL, SST<TurbSimLevel::RANS>>;
template
struct Driver<MixtureModel::FL, SST<TurbSimLevel::DES>>;

} // cfd