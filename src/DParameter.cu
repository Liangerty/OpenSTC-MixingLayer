#include "hip/hip_runtime.h"
#include "DParameter.cuh"
#include "ChemData.h"
#include "FlameletLib.cuh"
#include <filesystem>
#include <fstream>

cfd::DParameter::DParameter(cfd::Parameter &parameter, Species &species, Reaction *reaction,
                            FlameletLib *flamelet_lib) :
    myid{parameter.get_int("myid")}, dim{parameter.get_int("dimension")},
    problem_type{parameter.get_int("problem_type")}, inviscid_scheme{parameter.get_int("inviscid_scheme")},
    reconstruction{parameter.get_int("reconstruction")}, limiter{parameter.get_int("limiter")},
    entropy_fix_factor{parameter.get_real("entropy_fix_factor")},
    positive_preserving{parameter.get_bool("positive_preserving")},
    viscous_scheme{parameter.get_int("viscous_order")},
    perform_spanwise_average{parameter.get_bool("perform_spanwise_average")},
    rans_model{parameter.get_int("RANS_model")},
    turb_implicit{parameter.get_int("turb_implicit")}, n_var{parameter.get_int("n_var")},
    compressibility_correction{parameter.get_int("compressibility_correction")},
    chemSrcMethod{parameter.get_int("chemSrcMethod")}, n_scalar_transported{parameter.get_int("n_scalar_transported")},
    i_fl{parameter.get_int("i_fl")}, i_fl_cv{parameter.get_int("i_fl_cv")}, i_turb_cv{parameter.get_int("i_turb_cv")},
    Pr(parameter.get_real("prandtl_number")), cfl(parameter.get_real("cfl")),
    gradPInDiffusionFlux{parameter.get_bool("gradPInDiffusionFlux")},
    Prt(parameter.get_real("turbulent_prandtl_number")), Sct(parameter.get_real("turbulent_schmidt_number")),
    c_chi{parameter.get_real("c_chi")}, rho_ref{parameter.get_real("rho_inf")},
    a_ref2{parameter.get_real("speed_of_sound") * parameter.get_real("speed_of_sound")},
    v_ref{parameter.get_real("v_inf")}, T_ref{parameter.get_real("T_inf")},
    p_ref{parameter.get_real("p_inf")}, weno_eps_scale{
    parameter.get_real("rho_inf") * parameter.get_real("v_inf") * parameter.get_real("rho_inf") *
    parameter.get_real("v_inf")}, mach_ref{parameter.get_real("M_inf")},
    sponge_layer{parameter.get_bool("sponge_layer")},
    sponge_function{parameter.get_int("sponge_function")},
    sponge_iter{parameter.get_int("sponge_iter")}, spongeXPlusStart{parameter.get_real("spongeXPlusStart")},
    spongeXPlusEnd{parameter.get_real("spongeXPlusEnd")}, spongeXMinusStart{parameter.get_real("spongeXMinusStart")},
    spongeXMinusEnd{parameter.get_real("spongeXMinusEnd")}, spongeYPlusStart{parameter.get_real("spongeYPlusStart")},
    spongeYPlusEnd{parameter.get_real("spongeYPlusEnd")}, spongeYMinusStart{parameter.get_real("spongeYMinusStart")},
    spongeYMinusEnd{parameter.get_real("spongeYMinusEnd")}, spongeZPlusStart{parameter.get_real("spongeZPlusStart")},
    spongeZPlusEnd{parameter.get_real("spongeZPlusEnd")}, spongeZMinusStart{parameter.get_real("spongeZMinusStart")},
    spongeZMinusEnd{parameter.get_real("spongeZMinusEnd")} {
  if (parameter.get_int("myid") == 0) {
    if (inviscid_scheme == 51 || inviscid_scheme == 52 || inviscid_scheme == 71 || inviscid_scheme == 72)
      printf("\t->-> %-20e : WENO scale factor\n", weno_eps_scale);
  }

  if (parameter.get_bool("fixed_time_step")) {
    dt = parameter.get_real("dt");
  }

  const auto &spec = species;
  n_spec = spec.n_spec;
  n_scalar = parameter.get_int("n_scalar");
  if (reaction != nullptr) {
    n_reac = reaction->n_reac;
  }
  n_ps = parameter.get_int("n_ps");
  i_ps = parameter.get_int("i_ps");
  i_ps_cv = parameter.get_int("i_ps_cv");
  if (n_ps > 0) {
    hipMalloc(&sc_ps, n_ps * sizeof(real));
    hipMemcpy(sc_ps, parameter.get_real_array("sc_passive_scalar").data(), n_ps * sizeof(real),
               hipMemcpyHostToDevice);
    hipMalloc(&sct_ps, n_ps * sizeof(real));
    hipMemcpy(sct_ps, parameter.get_real_array("sct_passive_scalar").data(), n_ps * sizeof(real),
               hipMemcpyHostToDevice);
  }

  // species info
  auto mem_sz = n_spec * sizeof(real);
  hipMalloc(&mw, mem_sz);
  hipMemcpy(mw, spec.mw.data(), mem_sz, hipMemcpyHostToDevice);
  #ifdef HighTempMultiPart
  hipMalloc(&n_temperature_range, n_spec * sizeof(int));
  hipMemcpy(n_temperature_range, spec.n_temperature_range.data(), n_spec * sizeof(int), hipMemcpyHostToDevice);
  int n_ranges = 2;
  for (int l = 0; l < n_spec; ++l) {
    n_ranges = std::max(spec.n_temperature_range[l], n_ranges);
  }
  temperature_cuts.init_with_size(n_spec, n_ranges + 1);
  hipMemcpy(temperature_cuts.data(), spec.temperature_range.data(), temperature_cuts.size() * sizeof(real),
             hipMemcpyHostToDevice);
  therm_poly_coeff.allocate_memory(7, n_ranges, n_spec, 0);
  hipMemcpy(therm_poly_coeff.data(), spec.therm_poly_coeff.data(), sizeof(real) * therm_poly_coeff.size(),
             hipMemcpyHostToDevice);
  #else
  high_temp_coeff.init_with_size(n_spec, 7);
  hipMemcpy(high_temp_coeff.data(), spec.high_temp_coeff.data(), high_temp_coeff.size() * sizeof(real),
             hipMemcpyHostToDevice);
  low_temp_coeff.init_with_size(n_spec, 7);
  hipMemcpy(low_temp_coeff.data(), spec.low_temp_coeff.data(), low_temp_coeff.size() * sizeof(real),
             hipMemcpyHostToDevice);
  hipMalloc(&t_low, mem_sz);
  hipMalloc(&t_mid, mem_sz);
  hipMalloc(&t_high, mem_sz);
  hipMemcpy(t_low, spec.t_low.data(), mem_sz, hipMemcpyHostToDevice);
  hipMemcpy(t_mid, spec.t_mid.data(), mem_sz, hipMemcpyHostToDevice);
  hipMemcpy(t_high, spec.t_high.data(), mem_sz, hipMemcpyHostToDevice);
  #endif
  hipMalloc(&geometry, mem_sz);
  hipMemcpy(geometry, spec.geometry.data(), mem_sz, hipMemcpyHostToDevice);
  hipMalloc(&LJ_potent_inv, mem_sz);
  hipMemcpy(LJ_potent_inv, spec.LJ_potent_inv.data(), mem_sz, hipMemcpyHostToDevice);
  hipMalloc(&vis_coeff, mem_sz);
  hipMemcpy(vis_coeff, spec.vis_coeff.data(), mem_sz, hipMemcpyHostToDevice);
  WjDivWi_to_One4th.init_with_size(n_spec, n_spec);
  hipMemcpy(WjDivWi_to_One4th.data(), spec.WjDivWi_to_One4th.data(), WjDivWi_to_One4th.size() * sizeof(real),
             hipMemcpyHostToDevice);
  sqrt_WiDivWjPl1Mul8.init_with_size(n_spec, n_spec);
  hipMemcpy(sqrt_WiDivWjPl1Mul8.data(), spec.sqrt_WiDivWjPl1Mul8.data(),
             sqrt_WiDivWjPl1Mul8.size() * sizeof(real), hipMemcpyHostToDevice);
  binary_diffusivity_coeff.init_with_size(n_spec, n_spec);
  hipMemcpy(binary_diffusivity_coeff.data(), spec.binary_diffusivity_coeff.data(),
             binary_diffusivity_coeff.size() * sizeof(real), hipMemcpyHostToDevice);
  kb_over_eps_jk.init_with_size(n_spec, n_spec);
  hipMemcpy(kb_over_eps_jk.data(), spec.kb_over_eps_jk.data(),
             kb_over_eps_jk.size() * sizeof(real), hipMemcpyHostToDevice);
  hipMalloc(&ZRotF298, mem_sz);
  hipMemcpy(ZRotF298, spec.ZRotF298.data(), mem_sz, hipMemcpyHostToDevice);
  Sc = parameter.get_real("schmidt_number");

  // reaction info
  if (n_reac > 0) {
    hipMalloc(&reac_type, n_reac * sizeof(int));
    hipMemcpy(reac_type, reaction->label.data(), n_reac * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc(&rev_type, n_reac * sizeof(int));
    hipMemcpy(rev_type, reaction->rev_type.data(), n_reac * sizeof(int), hipMemcpyHostToDevice);
    stoi_f.init_with_size(n_reac, n_spec);
    hipMemcpy(stoi_f.data(), reaction->stoi_f.data(), stoi_f.size() * sizeof(int), hipMemcpyHostToDevice);
    stoi_b.init_with_size(n_reac, n_spec);
    hipMemcpy(stoi_b.data(), reaction->stoi_b.data(), stoi_b.size() * sizeof(int), hipMemcpyHostToDevice);
    mem_sz = n_reac * sizeof(real);
    hipMalloc(&reac_order, n_reac * sizeof(int));
    hipMemcpy(reac_order, reaction->order.data(), n_reac * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc(&A, mem_sz);
    hipMemcpy(A, reaction->A.data(), mem_sz, hipMemcpyHostToDevice);
    hipMalloc(&b, mem_sz);
    hipMemcpy(b, reaction->b.data(), mem_sz, hipMemcpyHostToDevice);
    hipMalloc(&Ea, mem_sz);
    hipMemcpy(Ea, reaction->Ea.data(), mem_sz, hipMemcpyHostToDevice);
    hipMalloc(&A2, mem_sz);
    hipMemcpy(A2, reaction->A2.data(), mem_sz, hipMemcpyHostToDevice);
    hipMalloc(&b2, mem_sz);
    hipMemcpy(b2, reaction->b2.data(), mem_sz, hipMemcpyHostToDevice);
    hipMalloc(&Ea2, mem_sz);
    hipMemcpy(Ea2, reaction->Ea2.data(), mem_sz, hipMemcpyHostToDevice);
    third_body_coeff.init_with_size(n_reac, n_spec);
    hipMemcpy(third_body_coeff.data(), reaction->third_body_coeff.data(), third_body_coeff.size() * sizeof(real),
               hipMemcpyHostToDevice);
    hipMalloc(&troe_alpha, mem_sz);
    hipMemcpy(troe_alpha, reaction->troe_alpha.data(), mem_sz, hipMemcpyHostToDevice);
    hipMalloc(&troe_t3, mem_sz);
    hipMemcpy(troe_t3, reaction->troe_t3.data(), mem_sz, hipMemcpyHostToDevice);
    hipMalloc(&troe_t1, mem_sz);
    hipMemcpy(troe_t1, reaction->troe_t1.data(), mem_sz, hipMemcpyHostToDevice);
    hipMalloc(&troe_t2, mem_sz);
    hipMemcpy(troe_t2, reaction->troe_t2.data(), mem_sz, hipMemcpyHostToDevice);
  }

  if (flamelet_lib->n_z != 0) {
    n_z = flamelet_lib->n_z;
    n_zPrime = flamelet_lib->n_zPrime;
    n_chi = flamelet_lib->n_chi;

    mem_sz = (n_z + 1) * sizeof(real);
    hipMalloc(&mix_frac, mem_sz);
    hipMemcpy(mix_frac, flamelet_lib->z.data(), mem_sz, hipMemcpyHostToDevice);
    zPrime.init_with_size(n_zPrime + 1, n_z + 1);
    hipMemcpy(zPrime.data(), flamelet_lib->zPrime.data(), zPrime.size() * sizeof(real), hipMemcpyHostToDevice);
    chi_min.init_with_size(n_zPrime + 1, n_z + 1);
    hipMemcpy(chi_min.data(), flamelet_lib->chi_min.data(), chi_min.size() * sizeof(real), hipMemcpyHostToDevice);
    chi_max.init_with_size(n_zPrime + 1, n_z + 1);
    hipMemcpy(chi_max.data(), flamelet_lib->chi_max.data(), chi_max.size() * sizeof(real), hipMemcpyHostToDevice);
    chi_min_j.init_with_size(n_zPrime + 1, n_z + 1);
    hipMemcpy(chi_min_j.data(), flamelet_lib->chi_min_j.data(), chi_min_j.size() * sizeof(int),
               hipMemcpyHostToDevice);
    chi_max_j.init_with_size(n_zPrime + 1, n_z + 1);
    hipMemcpy(chi_max_j.data(), flamelet_lib->chi_max_j.data(), chi_max_j.size() * sizeof(int),
               hipMemcpyHostToDevice);

    chi_ave.allocate_memory(n_chi, n_zPrime + 1, n_z + 1, 0);
    hipMemcpy(chi_ave.data(), flamelet_lib->chi_ave.data(), sizeof(real) * chi_ave.size(), hipMemcpyHostToDevice);
    yk_lib.allocate_memory(n_spec, n_chi, n_zPrime + 1, n_z + 1, 0);
    hipMemcpy(yk_lib.data(), flamelet_lib->yk.data(), sizeof(real) * yk_lib.size() * (n_z + 1),
               hipMemcpyHostToDevice);
  }

  if (parameter.get_bool("if_collect_statistics")) {
    n_reyAve = static_cast<int>(parameter.get_int_array("reyAveVarIndex").size());
    hipMalloc(&reyAveVarIndex, n_reyAve * sizeof(int));
    hipMemcpy(reyAveVarIndex, parameter.get_int_array("reyAveVarIndex").data(), n_reyAve * sizeof(int),
               hipMemcpyHostToDevice);
    rho_p_correlation = parameter.get_bool("rho_p_correlation");
    stat_tke_budget = parameter.get_bool("stat_tke_budget");
    if (n_spec > 0) {
      stat_species_dissipation_rate = parameter.get_bool("stat_species_dissipation_rate");
      stat_species_velocity_correlation = parameter.get_bool("stat_species_velocity_correlation");
      n_species_stat = parameter.get_int("n_species_stat");
      hipMalloc(&specStatIndex, n_species_stat * sizeof(int));
      hipMemcpy(specStatIndex, parameter.get_int_array("species_stat_index").data(), n_species_stat * sizeof(int),
                 hipMemcpyHostToDevice);
    }
  }

  // If mixing layer and multi-component, we need the mixture fraction info.
  if (problem_type == 1 && n_spec > 0) {
    beta_diff_inv = parameter.get_real("beta_diff_inv");
    beta_o = parameter.get_real("beta_o");
    nuc_mwc_inv = parameter.get_real("nuc_mwc_inv");
    nuh_mwh_inv = parameter.get_real("nuh_mwh_inv");
    half_nuo_mwo_inv = parameter.get_real("half_nuo_mwo_inv");
  }

  // the following parameters have been computed in "write_reference_state".
  if (problem_type == 1) {
    if (int i = parameter.get_int("characteristic_velocity_ml"); i == 0) {
      v_char = parameter.get_real("convective_velocity");
    } else {
      v_char = parameter.get_real("DeltaU");
    }
    delta_u = parameter.get_real("DeltaU");
  } else {
    v_char = parameter.get_real("v_inf");
  }

  memset(limit_flow.ll, 0, sizeof(real) * LimitFlow::max_n_var);
  memset(limit_flow.ul, 0, sizeof(real) * LimitFlow::max_n_var);
  memset(limit_flow.sv_inf, 0, sizeof(real) * (MAX_SPEC_NUMBER + 2));
  // density limits
  limit_flow.ll[0] = 1e-6 * parameter.get_real("rho_inf");
  limit_flow.ul[0] = 1e+3 * parameter.get_real("rho_inf");
  for (int l = 1; l < 4; ++l) {
    real vRef{parameter.get_real("v_inf")};
    if (abs(vRef) < 1) {
      vRef = parameter.get_real("speed_of_sound");
    }
    limit_flow.ll[l] = -1000 * vRef;
    limit_flow.ul[l] = 1000 * vRef;
  }
  // pressure limits
  limit_flow.ll[4] = 1e-6 * parameter.get_real("p_inf");
  limit_flow.ul[4] = 1e+3 * parameter.get_real("p_inf");
  if (rans_model == 2) {
    // SST model
    limit_flow.ul[5] = std::numeric_limits<real>::max();
    limit_flow.ul[6] = std::numeric_limits<real>::max();
  }
  auto &sv_inf{parameter.get_real_array("sv_inf")};
  for (int l = 0; l < n_scalar; ++l) {
    limit_flow.sv_inf[l] = sv_inf[l];
  }

  if (parameter.get_bool("sponge_layer")) {
    spongeX = parameter.get_int("spongeX");
    spongeY = parameter.get_int("spongeY");
    spongeZ = parameter.get_int("spongeZ");
    if (parameter.get_int("n_scalar") > 0) {
      hipMalloc(&sponge_scalar_iter, n_scalar * sizeof(int));
      hipMemcpy(sponge_scalar_iter, parameter.get_int_array("sponge_scalar_iter").data(), n_scalar * sizeof(int),
                 hipMemcpyHostToDevice);
    }
    if (spongeX == 1 || spongeX == 3) {
      sponge_sigma0 = parameter.get_real("spongeCoefficient") * v_char / (spongeXMinusStart - spongeXMinusEnd);
      printf("sponge_sigma0=%e\n", sponge_sigma0);
    }
    if (spongeX == 2 || spongeX == 3) {
      sponge_sigma1 = parameter.get_real("spongeCoefficient") * v_char / (spongeXPlusEnd - spongeXPlusStart);
      printf("sponge_sigma1=%e\n", sponge_sigma1);
    }
    if (spongeY == 1 || spongeY == 3) {
      sponge_sigma2 = parameter.get_real("spongeCoefficient") * v_char / (spongeYMinusStart - spongeYMinusEnd);
      printf("sponge_sigma2=%e\n", sponge_sigma2);
    }
    if (spongeY == 2 || spongeY == 3) {
      sponge_sigma3 = parameter.get_real("spongeCoefficient") * v_char / (spongeYPlusEnd - spongeYPlusStart);
      printf("sponge_sigma3=%e\n", sponge_sigma3);
    }
    if (spongeZ == 1 || spongeZ == 3) {
      sponge_sigma4 = parameter.get_real("spongeCoefficient") * v_char / (spongeZMinusStart - spongeZMinusEnd);
      printf("sponge_sigma4=%e\n", sponge_sigma4);
    }
    if (spongeZ == 2 || spongeZ == 3) {
      sponge_sigma5 = parameter.get_real("spongeCoefficient") * v_char / (spongeZPlusEnd - spongeZPlusStart);
      printf("sponge_sigma5=%e\n", sponge_sigma5);
    }
  }
}

//cfd::DParameter::~DParameter() {
//  hipFree(mw);
//  hipFree(t_low);
//  hipFree(t_mid);
//  hipFree(t_high);
//  hipFree(LJ_potent_inv);
//  hipFree(vis_coeff);
//  hipFree(reac_type);
//  hipFree(reac_order);
//  hipFree(A);
//  hipFree(b);
//  hipFree(Ea);
//  hipFree(A2);
//  hipFree(b2);
//  hipFree(Ea2);
//  hipFree(troe_alpha);
//  hipFree(troe_t3);
//  hipFree(troe_t1);
//  hipFree(troe_t2);
//  hipFree(mix_frac);
//}
__global__ void cfd::update_dt_global(DParameter *param, real dt) {
  param->dt = dt;
}
