#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include "kernels.h"
#include "Parallel.h"
#include <cstdio>
#include "DParameter.cuh"

namespace cfd{
void setup_gpu_device(int n_proc, int myid) {
  int deviceCount{0};
  hipGetDeviceCount(&deviceCount);

  if (deviceCount < n_proc) {
    printf("Not enough GPU devices.\n"
           "We want %d GPUs but only %d GPUs are available.\n"
           " Stop computing.\n", n_proc, deviceCount);
    MpiParallel::exit();
  }

  hipDeviceProp_t prop{};
  hipGetDeviceProperties(&prop, myid);
  hipSetDevice(myid);
  printf("\tProcess %d will compute on device [[%s]].\n", myid, prop.name);
}

__global__ void modify_cfl(DParameter *param, real cfl) {
  param->cfl = cfl;
}
}
