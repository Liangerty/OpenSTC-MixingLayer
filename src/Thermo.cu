#include "hip/hip_runtime.h"
#include "Thermo.cuh"
#include "DParameter.cuh"
#include "Constants.h"
#include "ChemData.h"

__device__ void cfd::compute_enthalpy(real t, real *enthalpy, const cfd::DParameter *param) {
  const real t2{t * t}, t3{t2 * t}, t4{t3 * t}, t5{t4 * t};
  auto &coeff = param->therm_poly_coeff;
  for (int i = 0; i < param->n_spec; ++i) {
    if (t < param->temperature_cuts(i, 0)) {
      const real tt = param->temperature_cuts(i, 0);
      const real tt2 = tt * tt, tt3 = tt2 * tt, tt4 = tt3 * tt, tt5 = tt4 * tt;
      enthalpy[i] =
          coeff(0, 0, i) * tt + 0.5 * coeff(1, 0, i) * tt2 + coeff(2, 0, i) * tt3 / 3 + 0.25 * coeff(3, 0, i) * tt4 +
          0.2 * coeff(4, 0, i) * tt5 + coeff(5, 0, i);
      const real cp =
          coeff(0, 0, i) + coeff(1, 0, i) * tt + coeff(2, 0, i) * tt2 + coeff(3, 0, i) * tt3 + coeff(4, 0, i) * tt4;
      enthalpy[i] += cp * (t - tt); // Do a linear interpolation for enthalpy
    } else if (t > param->temperature_cuts(i, param->n_temperature_range[i])) {
      const real tt = param->temperature_cuts(i, param->n_temperature_range[i]);
      const real tt2 = tt * tt, tt3 = tt2 * tt, tt4 = tt3 * tt, tt5 = tt4 * tt;
      const auto j = param->n_temperature_range[i] - 1;
      enthalpy[i] =
          coeff(0, j, i) * tt + 0.5 * coeff(1, j, i) * tt2 + coeff(2, j, i) * tt3 / 3 + 0.25 * coeff(3, j, i) * tt4 +
          0.2 * coeff(4, j, i) * tt5 + coeff(5, j, i);
      const real cp =
          coeff(0, j, i) + coeff(1, j, i) * tt + coeff(2, j, i) * tt2 + coeff(3, j, i) * tt3 + coeff(4, j, i) * tt4;
      enthalpy[i] += cp * (t - tt); // Do a linear interpolation for enthalpy
    } else {
      for (integer j = 0; j < param->n_temperature_range[i]; ++j) {
        if (param->temperature_cuts(i, j) <= t && t <= param->temperature_cuts(i, j + 1)) {
          enthalpy[i] =
              coeff(0, j, i) * t + 0.5 * coeff(1, j, i) * t2 + coeff(2, j, i) * t3 / 3 + 0.25 * coeff(3, j, i) * t4 +
              0.2 * coeff(4, j, i) * t5 + coeff(5, j, i);
          break;
        }
      }
    }
    enthalpy[i] *= cfd::R_u / param->mw[i];
  }
}

__device__ void cfd::compute_enthalpy_and_cp(real t, real *enthalpy, real *cp, const DParameter *param) {
  const double t2{t * t}, t3{t2 * t}, t4{t3 * t}, t5{t4 * t};
  auto &coeff = param->therm_poly_coeff;
  for (int i = 0; i < param->n_spec; ++i) {
    if (t < param->temperature_cuts(i, 0)) {
      const real tt = param->temperature_cuts(i, 0);
      const real tt2 = tt * tt, tt3 = tt2 * tt, tt4 = tt3 * tt, tt5 = tt4 * tt;
      enthalpy[i] =
          coeff(0, 0, i) * tt + 0.5 * coeff(1, 0, i) * tt2 + coeff(2, 0, i) * tt3 / 3 + 0.25 * coeff(3, 0, i) * tt4 +
          0.2 * coeff(4, 0, i) * tt5 + coeff(5, 0, i);
      cp[i] = coeff(0, 0, i) + coeff(1, 0, i) * tt + coeff(2, 0, i) * tt2 + coeff(3, 0, i) * tt3 + coeff(4, 0, i) * tt4;
      enthalpy[i] += cp[i] * (t - tt); // Do a linear interpolation for enthalpy
    } else if (t > param->temperature_cuts(i, param->n_temperature_range[i])) {
      const real tt = param->temperature_cuts(i, param->n_temperature_range[i]);
      const real tt2 = tt * tt, tt3 = tt2 * tt, tt4 = tt3 * tt, tt5 = tt4 * tt;
      const auto j = param->n_temperature_range[i] - 1;
      enthalpy[i] =
          coeff(0, j, i) * tt + 0.5 * coeff(1, j, i) * tt2 + coeff(2, j, i) * tt3 / 3 + 0.25 * coeff(3, j, i) * tt4 +
          0.2 * coeff(4, j, i) * tt5 + coeff(5, j, i);
      cp[i] = coeff(0, j, i) + coeff(1, j, i) * tt + coeff(2, j, i) * tt2 + coeff(3, j, i) * tt3 + coeff(4, j, i) * tt4;
      enthalpy[i] += cp[i] * (t - tt); // Do a linear interpolation for enthalpy
    } else {
      for (integer j = 0; j < param->n_temperature_range[i]; ++j) {
        if (param->temperature_cuts(i, j) <= t && t <= param->temperature_cuts(i, j + 1)) {
          enthalpy[i] =
              coeff(0, j, i) * t + 0.5 * coeff(1, j, i) * t2 + coeff(2, j, i) * t3 / 3 + 0.25 * coeff(3, j, i) * t4 +
              0.2 * coeff(4, j, i) * t5 + coeff(5, j, i);
          cp[i] = coeff(0, j, i) + coeff(1, j, i) * t + coeff(2, j, i) * t2 + coeff(3, j, i) * t3 + coeff(4, j, i) * t4;
          break;
        }
      }
    }
    cp[i] *= R_u / param->mw[i];
    enthalpy[i] *= R_u / param->mw[i];
  }
}

__device__ void cfd::compute_cp(real t, real *cp, cfd::DParameter *param) {
  const real t2{t * t}, t3{t2 * t}, t4{t3 * t};
  auto &coeff = param->therm_poly_coeff;
  for (int i = 0; i < param->n_spec; ++i) {
    if (t < param->temperature_cuts(i, 0)) {
      const real tt = param->temperature_cuts(i, 0);
      const real tt2 = tt * tt, tt3 = tt2 * tt, tt4 = tt3 * tt;
      cp[i] = coeff(0, 0, i) + coeff(1, 0, i) * tt + coeff(2, 0, i) * tt2 + coeff(3, 0, i) * tt3 + coeff(4, 0, i) * tt4;
    } else if (t > param->temperature_cuts(i, param->n_temperature_range[i])) {
      const real tt = param->temperature_cuts(i, param->n_temperature_range[i]);
      const real tt2 = tt * tt, tt3 = tt2 * tt, tt4 = tt3 * tt;
      const auto j = param->n_temperature_range[i] - 1;
      cp[i] = coeff(0, j, i) + coeff(1, j, i) * tt + coeff(2, j, i) * tt2 + coeff(3, j, i) * tt3 + coeff(4, j, i) * tt4;
    } else {
      for (integer j = 0; j < param->n_temperature_range[i]; ++j) {
        if (param->temperature_cuts(i, j) <= t && t <= param->temperature_cuts(i, j + 1)) {
          cp[i] = coeff(0, j, i) + coeff(1, j, i) * t + coeff(2, j, i) * t2 + coeff(3, j, i) * t3 + coeff(4, j, i) * t4;
          break;
        }
      }
    }
    cp[i] *= R_u / param->mw[i];
  }
}

void cfd::compute_cp(real t, real *cp, const cfd::Species &species) {
  const real t2{t * t}, t3{t2 * t}, t4{t3 * t};
  auto &coeff = species.therm_poly_coeff;
  for (int i = 0; i < species.n_spec; ++i) {
    if (t < species.temperature_range(i, 0)) {
      const real tt = species.temperature_range(i, 0);
      const real tt2 = tt * tt, tt3 = tt2 * tt, tt4 = tt3 * tt;
      cp[i] = coeff(0, 0, i) + coeff(1, 0, i) * tt + coeff(2, 0, i) * tt2 + coeff(3, 0, i) * tt3 + coeff(4, 0, i) * tt4;
    } else if (t > species.temperature_range(i, species.n_temperature_range[i])) {
      const real tt = species.temperature_range(i, species.n_temperature_range[i]);
      const real tt2 = tt * tt, tt3 = tt2 * tt, tt4 = tt3 * tt;
      const auto j = species.n_temperature_range[i] - 1;
      cp[i] = coeff(0, j, i) + coeff(1, j, i) * tt + coeff(2, j, i) * tt2 + coeff(3, j, i) * tt3 + coeff(4, j, i) * tt4;
    } else {
      for (integer j = 0; j < species.n_temperature_range[i]; ++j) {
        if (species.temperature_range(i, j) <= t && t <= species.temperature_range(i, j + 1)) {
          cp[i] = coeff(0, j, i) + coeff(1, j, i) * t + coeff(2, j, i) * t2 + coeff(3, j, i) * t3 + coeff(4, j, i) * t4;
          break;
        }
      }
    }
    cp[i] *= R_u / species.mw[i];
  }
}

__device__ void cfd::compute_gibbs_div_rt(real t, const cfd::DParameter *param, real *gibbs_rt) {
  const real t2{t * t}, t3{t2 * t}, t4{t3 * t}, t_inv{1 / t}, log_t{std::log(t)};
  auto &coeff = param->therm_poly_coeff;
  for (int i = 0; i < param->n_spec; ++i) {
    if (t < param->temperature_cuts(i, 0)) {
      const real tt = param->temperature_cuts(i, 0);
      const real tt2 = tt * tt, tt3 = tt2 * tt, tt4 = tt3 * tt, tt_inv = 1 / tt, log_tt = std::log(tt);
      gibbs_rt[i] = coeff(0, 0, i) * (1.0 - log_tt) - 0.5 * coeff(1, 0, i) * tt - coeff(2, 0, i) * tt2 / 6.0 -
                    coeff(3, 0, i) * tt3 / 12.0 - coeff(4, 0, i) * tt4 * 0.05 + coeff(5, 0, i) * tt_inv -
                    coeff(6, 0, i);
    } else if (t > param->temperature_cuts(i, param->n_temperature_range[i])) {
      const real tt = param->temperature_cuts(i, param->n_temperature_range[i]);
      const real tt2 = tt * tt, tt3 = tt2 * tt, tt4 = tt3 * tt, tt_inv = 1 / tt, log_tt = std::log(tt);
      const auto j = param->n_temperature_range[i] - 1;
      gibbs_rt[i] = coeff(0, j, i) * (1.0 - log_tt) - 0.5 * coeff(1, j, i) * tt - coeff(2, j, i) * tt2 / 6.0 -
                    coeff(3, j, i) * tt3 / 12.0 - coeff(4, j, i) * tt4 * 0.05 + coeff(5, j, i) * tt_inv -
                    coeff(6, j, i);
    } else {
      for (integer j = 0; j < param->n_temperature_range[i]; ++j) {
        if (param->temperature_cuts(i, j) <= t && t <= param->temperature_cuts(i, j + 1)) {
          gibbs_rt[i] = coeff(0, j, i) * (1.0 - log_t) - 0.5 * coeff(1, j, i) * t - coeff(2, j, i) * t2 / 6.0 -
                        coeff(3, j, i) * t3 / 12.0 - coeff(4, j, i) * t4 * 0.05 + coeff(5, j, i) * t_inv -
                        coeff(6, j, i);
          break;
        }
      }
    }
  }
}
