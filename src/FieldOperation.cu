#include "hip/hip_runtime.h"
#include "FieldOperation.cuh"

__device__ void
cfd::compute_temperature_and_pressure(int i, int j, int k, const DParameter *param, DZone *zone, real total_energy) {
  const int n_spec = param->n_spec;
  auto &Y = zone->sv;
  auto &bv = zone->bv;

  real mw{0};
  for (int l = 0; l < n_spec; ++l) {
    mw += Y(i, j, k, l) / param->mw[l];
  }
  mw = 1 / mw;
  const real gas_const = R_u / mw;
  const real e =
      total_energy / bv(i, j, k, 0) - 0.5 * (bv(i, j, k, 1) * bv(i, j, k, 1) + bv(i, j, k, 2) * bv(i, j, k, 2) +
                                             bv(i, j, k, 3) * bv(i, j, k, 3));

  real err{1}, t{bv(i, j, k, 5)};
  constexpr int max_iter{1000};
  constexpr real eps{1e-3};
  int iter = 0;

  real h_i[MAX_SPEC_NUMBER], cp_i[MAX_SPEC_NUMBER];
  while (err > eps && iter++ < max_iter) {
    compute_enthalpy_and_cp(t, h_i, cp_i, param);
    real cp_tot{0}, h{0};
    for (int l = 0; l < n_spec; ++l) {
      cp_tot += cp_i[l] * Y(i, j, k, l);
      h += h_i[l] * Y(i, j, k, l);
    }
    const real e_t = h - gas_const * t;
    const real cv = cp_tot - gas_const;
    const real t1 = t - (e_t - e) / cv;
    err = std::abs(1 - t1 / t);
    t = t1;
  }
  bv(i, j, k, 5) = t;
  bv(i, j, k, 4) = bv(i, j, k, 0) * t * gas_const;
}

__global__ void cfd::compute_velocity(cfd::DZone *zone) {
  const int ngg{zone->ngg}, mx{zone->mx}, my{zone->my}, mz{zone->mz};
  int i = (int) (blockDim.x * blockIdx.x + threadIdx.x) - ngg;
  int j = (int) (blockDim.y * blockIdx.y + threadIdx.y) - ngg;
  int k = (int) (blockDim.z * blockIdx.z + threadIdx.z) - ngg;
  if (i >= mx + ngg || j >= my + ngg || k >= mz + ngg) return;

  const auto &bv = zone->bv;

  zone->vel(i, j, k) = std::sqrt(
      bv(i, j, k, 1) * bv(i, j, k, 1) + bv(i, j, k, 2) * bv(i, j, k, 2) + bv(i, j, k, 3) * bv(i, j, k, 3));
}
