#include "hip/hip_runtime.h"
#include "Residual.cuh"

namespace cfd {
void steady_screen_output(int step, real err_max, gxl::Time &time, const std::array<real, 4> &res) {
  time.get_elapsed_time();
  FILE *history = std::fopen("history.dat", "a");
  fprintf(history, "%d\t%11.4e\n", step, err_max);
  fclose(history);

  printf("\n%38s    converged to: %11.4e\n", "rho", res[0]);
  printf("  n=%8d,                       V     converged to: %11.4e   \n", step, res[1]);
  printf("  n=%8d,                       p     converged to: %11.4e   \n", step, res[2]);
  printf("%38s    converged to: %11.4e\n", "T ", res[3]);
  printf("CPU time for this step is %16.8fs\n", time.step_time);
  printf("Total elapsed CPU time is %16.8fs\n", time.elapsed_time);
}

void
unsteady_screen_output(int step, real err_max, gxl::Time &time, const std::array<real, 4> &res, real dt,
                       real solution_time) {
  time.get_elapsed_time();
  FILE *history = std::fopen("history.dat", "a");
  fprintf(history, "%d\t%11.4e\n", step, err_max);
  fclose(history);

  printf("\n%38s    converged to: %11.4e\n", "rho", res[0]);
  printf("  n=%8d,   dt=%13.7e,   V     converged to: %11.4e   \n", step, dt, res[1]);
  printf("  n=%8d,   dt=%13.7e,   p     converged to: %11.4e   \n", step, dt, res[2]);
  printf("%38s    converged to: %11.4e\n", "T ", res[3]);
  printf("Current physical  time is %16.8es\n", solution_time);
  printf("CPU time for this step is %16.8fs\n", time.step_time);
  printf("Total elapsed CPU time is %16.8fs\n", time.elapsed_time);
}

__global__ void check_nan(DZone *zone, int blk, int myid) {
  const int mx{zone->mx}, my{zone->my}, mz{zone->mz};
  const int i = static_cast<int>(blockDim.x * blockIdx.x + threadIdx.x);
  const int j = static_cast<int>(blockDim.y * blockIdx.y + threadIdx.y);
  const int k = static_cast<int>(blockDim.z * blockIdx.z + threadIdx.z);
  if (i >= mx || j >= my || k >= mz) return;

  auto &bv = zone->bv;

  if (isnan(bv(i, j, k, 0)) || isnan(bv(i, j, k, 1)) || isnan(bv(i, j, k, 2)) || isnan(bv(i, j, k, 3)) ||
      isnan(bv(i, j, k, 4)) || isnan(bv(i, j, k, 5))) {
    printf("Proc %d, block %d, (%d, %d, %d), bv = {%e, %e, %e, %e, %e, %e}.\n", myid, blk, i, j, k, bv(i, j, k, 0),
           bv(i, j, k, 1), bv(i, j, k, 2), bv(i, j, k, 3), bv(i, j, k, 4), bv(i, j, k, 5));
  }
}
} // cfd
